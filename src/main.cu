#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <chrono>
#include "../headers/main.h"
#include "../headers/Simulation.h"
#include "../headers/Output.h"
#define TIMESTEP_SIZE 0.00005f

int main() {  

    // Allocate Simulation config
    Simulation* simulation;// = static_cast<Simulation*>(malloc(sizeof(Simulation)));
    hipMallocManaged(&simulation, sizeof(Simulation));
    //hipMemset(simulation, 0,sizeof(Simulation));

    // The sim is configured below:

    // How long the sim runs for (seconds)
    simulation->simTime = 5.0f;
    // Timestep size of the sim
    simulation->timestepSize = TIMESTEP_SIZE;
    // Logging samples per second
    simulation->outputFrameRate = 30;
    // The acceleration of the tank along the x axis
    simulation->accelerationX = 10.0f;
    // The acceleration of the tank along the y axis
    simulation->accelerationY = -10.0f;
    // How many particles to simulate
    simulation->numParticles = 500;
    // Size of each particle
    simulation->radius = 1.0f;

    // Setup simulation
    ParticleData* particleData = setupSimulation(simulation);

    // Setup simulation logger
    Output *logger = new Output(simulation);

    // Simulation tracking variables
    int stepCounter = 0;
    double currentTime = 0.0f;
    int sampleEvery = ( ( static_cast<int> (1/simulation->timestepSize)) / simulation->outputFrameRate);
   
    // Measure loop time
    auto StartTime = std::chrono::high_resolution_clock::now();

    while (currentTime < simulation->simTime) {
        // Step the simulation
        stepSimulation(simulation, particleData);

        // log
        if (stepCounter % sampleEvery == 0){
            logger->log_visualise(particleData);
            //logger->log_cg(sim);
        }

        // Print progress
        

        // increment simulation progress
        currentTime += TIMESTEP_SIZE;
        stepCounter++;
    }
    
    // Sim complete
    auto FinishTime = std::chrono::high_resolution_clock::now();
    auto TotalTime = std::chrono::duration_cast<std::chrono::microseconds>(FinishTime - StartTime);
    std::cout << "Main loop completed in: " << TotalTime.count() << std::endl;

    cleanupSimulation(simulation, particleData);
    return 0;
}
