#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "../headers/Simulation.h"
#include "../headers/Output.h"

int main() {  

    // Allocate Simulation config
    Simulation* simulation = static_cast<Simulation*>(malloc(sizeof(Simulation)));

    // The sim is configured below:

    // How long the sim runs for (seconds)
    simulation->simTime = 5.0f;
    // Timestep size of the sim
    simulation->timestepSize = 0.00005f;
    // Logging samples per second
    simulation->outputFrameRate = 30;
    // The acceleration of the tank along the x axis
    simulation->accelerationX = 10.0f;
    // The acceleration of the tank along the y axis
    simulation->accelerationY = -10.0f;
    // How many particles to simulate
    simulation->numParticles = 500;
    // Size of each particle
    simulation->radius = 1.0f;

    // Setup simulation
    ParticleData* particleData = setupSimulation(simulation);

    // Setup simulation logger
    Output *logger = new Output(simulation);

    // Simulation tracking variables
    int stepCounter = 0;
    double currentTime = 0.0f;
    int sampleEvery = ( ( static_cast<int> (1/simulation->timestepSize)) / simulation->outputFrameRate);
    
    while (currentTime < simulation->simTime) {
        // Step the simulation
        stepSimulation(simulation, particleData);

        // log
        if (stepCounter % sampleEvery == 0){
            logger->log_visualise(particleData);
            //logger->log_cg(sim);
        }


        // Print progress

        // increment simulation progress
        currentTime += simulation->timestepSize;
        stepCounter++;
    }
    

    /*
    // Sample rate for logging
    int sampleEvery = ( ( static_cast<int> (1/TIMESTEP_SIZE)) / OUTPUT_FRAME_RATE);

    // Setup sim
    Simulation *sim = new Simulation(NUMBER_OF_PARTICLES, TIMESTEP_SIZE, ACCELERATION_X, ACCELERATION_Y);
    Output *logger = new Output();

    // run sim main
    int stepCounter = 0;

    // Sim loop
    double currentTime = 0.0f;
    while (currentTime < SIM_TIME) {
        
        // run simulation
        sim->step();

        // If this step in the sim is a sample step, sample it.
        //if (stepCounter % sampleEvery == 0){
        //    logger->log_visualise(sim);
        //    logger->log_cg(sim);
        //}

        // Every second print simulation time that has elapsed.
        if (stepCounter % (sampleEvery*OUTPUT_FRAME_RATE) == 0){
            std::cout << "Completed " << currentTime << "s" << std::endl;
        }

        // increment current time
        currentTime += TIMESTEP_SIZE;
        stepCounter++;
    }

    */
    // Sim complete
    std::cout << "Simulation complete" << std::endl;

    cleanupSimulation(simulation, particleData);
    return 0;
}
