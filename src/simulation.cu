#include "hip/hip_runtime.h"
#include "../headers/Simulation.h"

// Initialise the simulation and its variables
ParticleData* setupSimulation(Simulation* simulation) {
    // Allocate a simulation structure
    ParticleData* particleData = static_cast<ParticleData*>(malloc(sizeof(ParticleData)));

    // Allocate vectors that hold particle data
    particleData->positionX = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
    particleData->positionY = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
    particleData->velocityX = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
    particleData->velocityY = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
    particleData->accelerationX = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
    particleData->accelerationY = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));

    // Initialise the particles locations
    srand(1);
    for (int i = 0; i < simulation->numParticles; i++) {
        // Give each particle an initial random X and Y location.
        particleData->positionX[i] = (static_cast <double> (rand()) / (static_cast <double> (RAND_MAX/90.0f))) + 5.0f;
        particleData->positionY[i] = (static_cast <double> (rand()) / (static_cast <double> (RAND_MAX/90.0f))) + 5.0f;
    }

    return particleData;
}

// Step the simulation one step
void stepSimulation(Simulation* simulation, ParticleData* particleData) {
    // Step Particle velocities
    stepVelocities(simulation, particleData);

    // resolve collisions
    collisions(simulation, particleData);

    // Step positions
    stepPositions(simulation, particleData);
}

void stepVelocities(Simulation* simulation, ParticleData* particleData) {
    // Can be vectorised
    for (int i = 0; i < simulation->numParticles; i++) {
        particleData->velocityX[i] = particleData->velocityX[i] + simulation->accelerationX * simulation->timestepSize;
        particleData->velocityY[i] = particleData->velocityY[i] + simulation->accelerationY * simulation->timestepSize;
    }
}

void stepPositions(Simulation* simulation, ParticleData* particleData) {
    // Can be vectorised
    for (int i = 0; i < simulation->numParticles; i++) {
        particleData->positionX[i] = particleData->positionX[i] + particleData->velocityY[i] * simulation->timestepSize;
        particleData->positionY[i] = particleData->positionY[i] + particleData->velocityY[i] * simulation->timestepSize;
    }
}

void collisions(Simulation* simulation, ParticleData* particleData) {

    // fluid (particle-particle) collisions    
    for (int i = 0; i < simulation->numParticles; i++) {
        for (int j = i+1; j < simulation->numParticles; j++) {
            double radius = simulation->radius;

            // Pythag
            double distance = sqrt(pow(particleData->positionX[i]-particleData->positionX[j], 2) + pow(particleData->positionY[i]-particleData->positionY[j], 2));

            // 1. is there a collision?
            if (distance < 2*radius) {
                // Calculate the Normal
                double nx = (particleData->positionX[j] - particleData->positionX[i]) / distance;
                double ny = (particleData->positionY[j] - particleData->positionY[i]) / distance;

                // inner product the normal and velocity
                double innerNormalI = particleData->velocityX[i] * nx + particleData->velocityY[i] * ny;
                double innerNormalJ = particleData->velocityX[j] * nx + particleData->velocityY[j] * ny;


                // Calculate the tangent
                double tx = -ny;
                double ty = nx;

                // inner product the tangent and velocity
                double innerTangentI = particleData->velocityX[i] * tx + particleData->velocityY[i] * ty;
                double innerTangentJ = particleData->velocityX[j] * tx + particleData->velocityY[j] * ty;

                // Conservation of momentum in 1D
                // Let mass = 1
                // momentum one and two end up being equal to innerNormal1 and innerNormal2 because m1=m2

                // Update the velocities
                particleData->velocityX[i] = (tx * innerTangentI + nx * innerNormalI);
                particleData->velocityY[i] = (ty * innerTangentI + ny * innerNormalI);
                particleData->velocityX[j] = (tx * innerTangentJ + nx * innerNormalJ);
                particleData->velocityY[j] = (ty * innerTangentJ + ny * innerNormalJ);


                // Move so not overlapping
                double overlap = 0.5f * (distance - (2.0f*radius));

                particleData->positionX[i] -= overlap * (particleData->positionX[i] - particleData->positionX[j])/distance;
                particleData->positionY[i] -= overlap * (particleData->positionY[i] - particleData->positionY[j])/distance;

                particleData->positionX[j] += overlap * (particleData->positionX[i] - particleData->positionX[j])/distance;
                particleData->positionY[j] += overlap * (particleData->positionY[i] - particleData->positionY[j])/distance;
            }
        }
    }



    // Evaluate wall conditions
    for (int i = 0; i < simulation->numParticles; i++) {

        // x dimension (can be its own kernel)
        if (particleData->positionX[i] - simulation->radius < 0.0f){

            particleData->positionX[i] = 0.0f + simulation->radius;
            particleData->velocityX[i] = 0.0f;
        }
        if (particleData->positionX[i] + simulation->radius > 100.0f){

            particleData->positionX[i] = 100.0f - simulation->radius;
            particleData->velocityX[i] = 0.0f;

        }

        // y dimension (can be its own kernel)
        if (particleData->positionY[i] - simulation->radius < 0.0f){

            particleData->positionY[i] = 0.0f + simulation->radius;
            particleData->velocityY[i] = 0.0f;
        }
        if (particleData->positionY[i] + simulation->radius > 100.0f){

            particleData->positionY[i] = 100.0f - simulation->radius;
            particleData->velocityY[i] = 0.0f;

        }
    }
}

void cleanupSimulation(Simulation* simulation, ParticleData* particleData) {
    // Free the simulation
    free(simulation);

    // Free the particle data
    free(particleData->positionX);
    free(particleData->positionY);
    free(particleData->velocityX);
    free(particleData->velocityY);
    free(particleData->accelerationX);
    free(particleData->accelerationY);
    free(particleData);
}
