#include "hip/hip_runtime.h"
#include "../headers/Simulation.h"
#include "../headers/particleCollision.h"

// Initialise the simulation and its variables
ParticleData *setupSimulation(Simulation *simulation) {
  // Allocate a simulation structure
  ParticleData *particleData;

  // Allocate memory for the particle data struct
  hipMallocManaged(&particleData, sizeof(ParticleData));

  // Allocate memory for the X positions of the particles
  hipMallocManaged(&(particleData->positionX),
                    simulation->numParticles * sizeof(double));

  // Allocate memory for the Y positions of the particles
  hipMallocManaged(&(particleData->positionY),
                    simulation->numParticles * sizeof(double));

  // Allocate memory for the X-axis velocities of particles
  hipMallocManaged(&(particleData->velocityX),
                    simulation->numParticles * sizeof(double));

  // Allocate memory for the y-axis velocities of particles
  hipMallocManaged(&(particleData->velocityY),
                    simulation->numParticles * sizeof(double));

  // Give each particle an initial X and Y location.
  srand(1);
  for (int i = 0; i < simulation->numParticles; i++) {
    particleData->positionX[i] = (static_cast<double>(rand()) /
                                  (static_cast<double>(RAND_MAX / 90.0f))) +
                                 5.0f;
    particleData->positionY[i] = (static_cast<double>(rand()) /
                                  (static_cast<double>(RAND_MAX / 90.0f))) +
                                 5.0f;
  }

  return particleData;
}

// Step the simulation one step
void stepSimulation(Simulation *simulation, ParticleData *particleData) {
  // Step Particle velocities

  // How many blocks to run kernel over:
  int blocks = (simulation->numParticles / 1024) + 1;
  // Evenly distribute the number of threads over the blocks for peak
  // performance
  int threads = (simulation->numParticles / blocks) + 1;

  // Handle particle-particle collisions
  particleCollisions(simulation, particleData);

  // Resolve particle-wall collisions
  wallCollisions<<<blocks, threads>>>(simulation, particleData);

  // Step velocities
  stepVelocities<<<blocks, threads>>>(simulation, particleData);

  // Step positions
  stepPositions<<<blocks, threads>>>(simulation, particleData);
}

// Steps the velocities of the particles one timestep
__global__ void stepVelocities(Simulation *simulation,
                               ParticleData *particleData) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < simulation->numParticles) {
    particleData->velocityX[i] =
        particleData->velocityX[i] +
        simulation->accelerationX * simulation->timestepSize;
    particleData->velocityY[i] =
        particleData->velocityY[i] +
        simulation->accelerationY * simulation->timestepSize;
  }
}

// Steps the positions of the particles one timestep
__global__ void stepPositions(Simulation *simulation,
                              ParticleData *particleData) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < simulation->numParticles) {
    particleData->positionX[i] =
        particleData->positionX[i] +
        particleData->velocityX[i] * simulation->timestepSize;
    particleData->positionY[i] =
        particleData->positionY[i] +
        particleData->velocityY[i] * simulation->timestepSize;
  }
}

// Resolves particle-wall collisions
__global__ void wallCollisions(Simulation *simulation,
                               ParticleData *particleData) {
  
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (i < simulation->numParticles) {
    
    // Resolve any collisions with the horizontal walls (x axis)
    if (particleData->positionX[i] - simulation->radius < 0.0f) {

      particleData->positionX[i] = 0.0f + simulation->radius;
      particleData->velocityX[i] = 0.0f;
    }
    if (particleData->positionX[i] + simulation->radius > 100.0f) {

      particleData->positionX[i] = 100.0f - simulation->radius;
      particleData->velocityX[i] = 0.0f;
    }

    // Resolve any collisions with the vertical walls (y axis)
    if (particleData->positionY[i] - simulation->radius < 0.0f) {

      particleData->positionY[i] = 0.0f + simulation->radius;
      particleData->velocityY[i] = 0.0f;
    }
    if (particleData->positionY[i] + simulation->radius > 100.0f) {

      particleData->positionY[i] = 100.0f - simulation->radius;
      particleData->velocityY[i] = 0.0f;
    }
  }
}

// Frees (host and CUDA) memory used by the simulation.
void cleanupSimulation(Simulation *simulation, ParticleData *particleData) {
  // Free the simulation
  hipFree(simulation);

  // Free the particle data
  hipFree(particleData->positionX);
  hipFree(particleData->positionY);
  hipFree(particleData->velocityX);
  hipFree(particleData->velocityY);
  hipFree(particleData->accelerationX);
  hipFree(particleData->accelerationY);
  hipFree(particleData);
}
