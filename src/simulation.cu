#include "hip/hip_runtime.h"
#include "../headers/Simulation.h"

// Initialise the simulation and its variables
ParticleData* setupSimulation(Simulation* simulation) {
    // Allocate a simulation structure
    ParticleData* particleData;
    hipMallocManaged( &particleData, sizeof(ParticleData));

    // Allocate vectors that hold particle data
    /*particleData->positionX = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
      particleData->positionY = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
      particleData->velocityX = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
      particleData->velocityY = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
      particleData->accelerationX = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));
      particleData->accelerationY = static_cast<double*>(calloc(simulation->numParticles, sizeof(double)));*/
    hipMallocManaged(&(particleData->positionX), simulation->numParticles*sizeof(double));
    hipMallocManaged(&(particleData->positionY), simulation->numParticles*sizeof(double));
    hipMallocManaged(&(particleData->velocityX), simulation->numParticles*sizeof(double));
    hipMallocManaged(&(particleData->velocityY), simulation->numParticles*sizeof(double));

    // Initialise the particles locations
    srand(1);
    for (int i = 0; i < simulation->numParticles; i++) {
        // Give each particle an initial random X and Y location.
        particleData->positionX[i] = (static_cast <double> (rand()) / (static_cast <double> (RAND_MAX/90.0f))) + 5.0f;
        particleData->positionY[i] = (static_cast <double> (rand()) / (static_cast <double> (RAND_MAX/90.0f))) + 5.0f;
    }

    return particleData;
}

// Step the simulation one step
void stepSimulation(Simulation* simulation, ParticleData* particleData) {
    // Step Particle velocities

    // How many blocks to run kernel over:
    int blocks = (simulation->numParticles/1024) + 1;
    // Evenly distribute the number of threads over the blocks for peak performance
    int threads = (simulation->numParticles/blocks) + 1;

    stepVelocities<<<blocks, threads>>>(simulation, particleData);
    hipDeviceSynchronize();

    particleCollisions(simulation, particleData, distances);

    // Resolve particle-wall collisions
    wallCollisions<<<blocks, threads>>>(simulation, particleData);
    hipDeviceSynchronize();

    // Step positions
    stepPositions<<<blocks, threads>>>(simulation, particleData);
    hipDeviceSynchronize();
}

__global__
void stepVelocities(Simulation* simulation, ParticleData* particleData) {
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int stride = blockDim.x * gridDim.x;
    //for (int i = index; i < simulation->numParticles; i += stride) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < simulation->numParticles) { 
        particleData->velocityX[i] = particleData->velocityX[i] + simulation->accelerationX * simulation->timestepSize;
        particleData->velocityY[i] = particleData->velocityY[i] + simulation->accelerationY * simulation->timestepSize;
    }
}

__global__
void stepPositions(Simulation* simulation, ParticleData* particleData) {
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int stride = blockDim.x * gridDim.x;
    //for (int i = index; i < simulation->numParticles; i += stride) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < simulation->numParticles) { 
        particleData->positionX[i] = particleData->positionX[i] + particleData->velocityY[i] * simulation->timestepSize;
        particleData->positionY[i] = particleData->positionY[i] + particleData->velocityY[i] * simulation->timestepSize;
    }
}

void particleCollisions(Simulation* simulation, ParticleData* particleData, double* distances) {
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int stride = blockDim.x * gridDim.x;
    
    // fluid (particle-particle) collisions    
    for (int i = 0; i < simulation->numParticles; i += 1) {
        for (int j = i+1; j < simulation->numParticles; j++) {
            double radius = simulation->radius;

            // Pythag
            //double distance = sqrt(pow(particleData->positionX[i]-particleData->positionX[j], 2) 
            //+ pow(particleData->positionY[i]-particleData->positionY[j], 2));

            double distance = distances[i * simulation->numParticles + j];

            // 1. is there a collision?
            // Calculate the Normal
            double nx = (particleData->positionX[j] - particleData->positionX[i]) / distance;
            double ny = (particleData->positionY[j] - particleData->positionY[i]) / distance;

            // inner product the normal and velocity
            double innerNormalI = particleData->velocityX[i] * nx + particleData->velocityY[i] * ny;
            double innerNormalJ = particleData->velocityX[j] * nx + particleData->velocityY[j] * ny;


            // Calculate the tangent
            double tx = -ny;
            double ty = nx;

            // inner product the tangent and velocity
            double innerTangentI = particleData->velocityX[i] * tx + particleData->velocityY[i] * ty;
            double innerTangentJ = particleData->velocityX[j] * tx + particleData->velocityY[j] * ty;

            double overlap = 0.5f * (distance - (2.0f*radius));

            double newPositionXI = particleData->positionX[i] - overlap * (particleData->positionX[i] - particleData->positionX[j])/distance;
            double newPositionYI = particleData->positionY[i] - overlap * (particleData->positionY[i] - particleData->positionY[j])/distance;
            double newPositionXJ = particleData->positionX[j] + overlap * (particleData->positionX[i] - particleData->positionX[j])/distance;
            double newPositionYJ = particleData->positionY[j] + overlap * (particleData->positionY[i] - particleData->positionY[j])/distance;

            // Update the velocities
            if (distance < 2*radius) {
                // 1D conservation of momentum
                particleData->velocityX[i] = (tx * innerTangentI + nx * innerNormalI);
                particleData->velocityY[i] = (ty * innerTangentI + ny * innerNormalI);
                particleData->velocityX[j] = (tx * innerTangentJ + nx * innerNormalJ);
                particleData->velocityY[j] = (ty * innerTangentJ + ny * innerNormalJ);


                // Move so not overlapping
                particleData->positionX[i] = newPositionXI;
                particleData->positionY[i] = newPositionYI;

                particleData->positionX[j] = newPositionXJ;
                particleData->positionY[j] = newPositionYJ;
            }
        }
    }

}

__global__
void wallCollisions(Simulation* simulation, ParticleData* particleData) {
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int stride = blockDim.x * gridDim.x;

    // Evaluate wall conditions
    //for (int i = index; i < simulation->numParticles; i += stride) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < simulation->numParticles) { 
        // x dimension (can be its own kernel)
        if (particleData->positionX[i] - simulation->radius < 0.0f){

            particleData->positionX[i] = 0.0f + simulation->radius;
            particleData->velocityX[i] = 0.0f;
        }
        if (particleData->positionX[i] + simulation->radius > 100.0f){

            particleData->positionX[i] = 100.0f - simulation->radius;
            particleData->velocityX[i] = 0.0f;

        }

        // y dimension (can be its own kernel)
        if (particleData->positionY[i] - simulation->radius < 0.0f){

            particleData->positionY[i] = 0.0f + simulation->radius;
            particleData->velocityY[i] = 0.0f;
        }
        if (particleData->positionY[i] + simulation->radius > 100.0f){

            particleData->positionY[i] = 100.0f - simulation->radius;
            particleData->velocityY[i] = 0.0f;

        }
    }

}

void cleanupSimulation(Simulation* simulation, ParticleData* particleData) {
    // Free the simulation
    hipFree(simulation);

    // Free the particle data
    hipFree(particleData->positionX);
    hipFree(particleData->positionY);
    hipFree(particleData->velocityX);
    hipFree(particleData->velocityY);
    hipFree(particleData->accelerationX);
    hipFree(particleData->accelerationY);
    hipFree(particleData);
}
