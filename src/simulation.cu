#include "hip/hip_runtime.h"
#include "../headers/Simulation.h"
#include "../headers/particleCollision.h"

// Initialise the simulation and its variables
ParticleData* setupSimulation(Simulation* simulation) {
    // Allocate a simulation structure
    ParticleData* particleData;

    hipMallocManaged( &particleData, sizeof(ParticleData));
    hipMallocManaged(&(particleData->positionX), simulation->numParticles*sizeof(double));
    hipMallocManaged(&(particleData->positionY), simulation->numParticles*sizeof(double));
    hipMallocManaged(&(particleData->velocityX), simulation->numParticles*sizeof(double));
    hipMallocManaged(&(particleData->velocityY), simulation->numParticles*sizeof(double));

    // Initialise the particles locations
    srand(1);
    for (int i = 0; i < simulation->numParticles; i++) {
        // Give each particle an initial random X and Y location.
        particleData->positionX[i] = (static_cast <double> (rand()) / (static_cast <double> (RAND_MAX/90.0f))) + 5.0f;
        particleData->positionY[i] = (static_cast <double> (rand()) / (static_cast <double> (RAND_MAX/90.0f))) + 5.0f;
    }

    return particleData;
}

// Step the simulation one step
void stepSimulation(Simulation* simulation, ParticleData* particleData) {
    // Step Particle velocities

    // How many blocks to run kernel over:
    int blocks = (simulation->numParticles/1024) + 1;
    // Evenly distribute the number of threads over the blocks for peak performance
    int threads = (simulation->numParticles/blocks) + 1;

    particleCollisions(simulation, particleData);

    // Resolve particle-wall collisions
    wallCollisions<<<blocks, threads>>>(simulation, particleData);

    stepVelocities<<<blocks, threads>>>(simulation, particleData);
    
    // Step positions
    stepPositions<<<blocks, threads>>>(simulation, particleData);
}

__global__
void stepVelocities(Simulation* simulation, ParticleData* particleData) {
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int stride = blockDim.x * gridDim.x;
    //for (int i = index; i < simulation->numParticles; i += stride) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < simulation->numParticles) { 
        particleData->velocityX[i] = particleData->velocityX[i] + simulation->accelerationX * simulation->timestepSize;
        particleData->velocityY[i] = particleData->velocityY[i] + simulation->accelerationY * simulation->timestepSize;
    }
}

__global__
void stepPositions(Simulation* simulation, ParticleData* particleData) {
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int stride = blockDim.x * gridDim.x;
    //for (int i = index; i < simulation->numParticles; i += stride) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < simulation->numParticles) { 
        particleData->positionX[i] = particleData->positionX[i] + particleData->velocityY[i] * simulation->timestepSize;
        particleData->positionY[i] = particleData->positionY[i] + particleData->velocityY[i] * simulation->timestepSize;
    }
}



__global__
void wallCollisions(Simulation* simulation, ParticleData* particleData) {
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int stride = blockDim.x * gridDim.x;

    // Evaluate wall conditions
    //for (int i = index; i < simulation->numParticles; i += stride) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < simulation->numParticles) { 
        // x dimension (can be its own kernel)
        if (particleData->positionX[i] - simulation->radius < 0.0f){

            particleData->positionX[i] = 0.0f + simulation->radius;
            particleData->velocityX[i] = 0.0f;
        }
        if (particleData->positionX[i] + simulation->radius > 100.0f){

            particleData->positionX[i] = 100.0f - simulation->radius;
            particleData->velocityX[i] = 0.0f;

        }

        // y dimension (can be its own kernel)
        if (particleData->positionY[i] - simulation->radius < 0.0f){

            particleData->positionY[i] = 0.0f + simulation->radius;
            particleData->velocityY[i] = 0.0f;
        }
        if (particleData->positionY[i] + simulation->radius > 100.0f){

            particleData->positionY[i] = 100.0f - simulation->radius;
            particleData->velocityY[i] = 0.0f;

        }
    }

}

void cleanupSimulation(Simulation* simulation, ParticleData* particleData) {
    // Free the simulation
    hipFree(simulation);

    // Free the particle data
    hipFree(particleData->positionX);
    hipFree(particleData->positionY);
    hipFree(particleData->velocityX);
    hipFree(particleData->velocityY);
    hipFree(particleData->accelerationX);
    hipFree(particleData->accelerationY);
    hipFree(particleData);
}
